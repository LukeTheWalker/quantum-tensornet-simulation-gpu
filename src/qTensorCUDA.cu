#include "hip/hip_runtime.h"
#include "qTensor.hpp"
#include "qTensorCUDA.cuh"
#include "bitsetCU.cuh"
#include "Contraction.hpp"

#include <hip/hip_complex.h>
#include <unordered_map>
#include <hipblas.h>

// using namespace cuda_classes;
using cpx = hipComplex;

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

auto findCommonValues = [](std::vector<unsigned char> set1, std::vector<unsigned char> set2) -> std::vector<unsigned char> {
    std::vector<unsigned char> commonValues;
    for (auto value : set1) {
        if (std::find(set2.begin(), set2.end(), value) != set2.end()) {
            commonValues.push_back((unsigned char)value);
        }
    }
    return commonValues;
};

__device__ void keepNtoMbits(cuda_classes::bitset& bits, int n, int m) 
{ 
    for (int i = 0; i < n; i++) 
    { 
        bits.set(i, 0);
    }  
    for (int i = m; i < 64; i++) 
    { 
        bits.set(i, 0);
    }  
}

__device__ unsigned char getIndexInSet(unsigned char* set, unsigned char element, int size) {
    for (int i = 0; i < size; i++) {
        if (set[i] == element) {
            return i;
        }
    }
    return 255; // Element not found in the set
}

__device__ void print_bitset(cuda_classes::bitset& bits) {
    for (int i = 0; i < 64; i++) {
        printf("%d", bits.get(i));
    }
    printf("\n");
}

__global__ void contractionKernel(unsigned char* d_spanA, unsigned char* d_spanB, unsigned char* d_newSpan, unsigned char* connections, cpx* d_valuesA, cpx* d_valuesB, cpx* d_resultValues, int rankA, int rankB, int rankResult, int connectionsSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= (1 << (rankResult*2))) return;

    cuda_classes::bitset bits(i);

    cuda_classes::bitset a(0);
    cuda_classes::bitset b(0);

    auto lane = d_newSpan;
    for (int k = 0 ; k < rankResult; k++)
    {
        unsigned char indexA = getIndexInSet(d_spanA, *lane, rankA);
        unsigned char indexB = getIndexInSet(d_spanB, *lane, rankB);

        if (indexA != 255) a.set(2*rankA - indexA - 1, bits.get(rankResult*2 - 1 - k));
        else               b.set(2*rankB - indexB - 1, bits.get(rankResult*2 - 1 - k));

        if (indexB != 255) b.set(rankB - indexB - 1, bits.get(rankResult - 1 - k));
        else               a.set(rankA - indexA - 1, bits.get(rankResult - 1 - k));

        lane++;
    }

    for (int m = 0; m < (1 << connectionsSize); m++)
    {
        cuda_classes::bitset address_vacant(m);
        int cnt = 0;
        for (int c = 0; c < connectionsSize; c++)
        {
            unsigned char indexA = getIndexInSet(d_spanA, connections[c], rankA);
            unsigned char indexB = getIndexInSet(d_spanB, connections[c], rankB);
            a.set(rankA - indexA - 1, address_vacant.get(cnt));
            b.set(2*rankB - indexB - 1, address_vacant.get(cnt));
            cnt++;
        }

        cpx value = hipCmulf(d_valuesA[a.to_ulong()], d_valuesB[b.to_ulong()]);
        d_resultValues[i] = hipCaddf(d_resultValues[i], value);
    }
}

int round_div_up (int a, int b){
    return (a + b - 1)/b;
}

QTensor contractionGPU(QTensor A, QTensor B) 
{
    std::set<unsigned char> newSpan;
    newSpan.insert(A.span.begin(), A.span.end());
    newSpan.insert(B.span.begin(), B.span.end());

    // convert all sets to vectors
    std::vector<unsigned char> newSpanVec(newSpan.begin(), newSpan.end());
    std::vector<unsigned char> spanA(A.span.begin(), A.span.end());
    std::vector<unsigned char> spanB(B.span.begin(), B.span.end());

    QTensor result = QTensor(newSpan);
    std::vector<std::complex<float>> resultValues(1 << (result.rank*2), {0.0, 0.0});

    std::vector<unsigned char> connections = findCommonValues(spanA, spanB);

    /** ----------------------------- CUDA ----------------------------- **/
    hipError_t err;

    // start transfering data to the GPU
    unsigned char* d_spanA, *d_spanB, *d_newSpan, *d_connections;
    cpx* d_valuesA, *d_valuesB, *d_resultValues;

    // memcopies
    {
        err = hipMalloc(&d_spanA, A.span.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc(&d_spanB, B.span.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc(&d_newSpan, newSpan.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc(&d_connections, connections.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMalloc(&d_valuesA, A.values.size() * sizeof(cpx)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc(&d_valuesB, B.values.size() * sizeof(cpx)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc(&d_resultValues, resultValues.size() * sizeof(cpx)); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemcpy(d_spanA, spanA.data(), A.span.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_spanB, spanB.data(), B.span.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_newSpan, newSpanVec.data(), newSpan.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_connections, connections.data(), connections.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemcpy(d_valuesA, A.values.data(), A.values.size() * sizeof(cpx), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_valuesB, B.values.data(), B.values.size() * sizeof(cpx), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_resultValues, resultValues.data(), resultValues.size() * sizeof(cpx), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    }

    // kernel call
    {
        int nels = 1 << (result.rank*2);
        int blocksize = 256;
        int numBlocks = round_div_up(nels, blocksize);

        // std::cout << "numBlocks: " << numBlocks << " blocksize: " << blocksize << std::endl;

        contractionKernel<<<numBlocks, blocksize>>>(d_spanA, d_spanB, d_newSpan, d_connections, d_valuesA, d_valuesB, d_resultValues, A.rank, B.rank, result.rank, connections.size());
        err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
        err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemcpy(resultValues.data(), d_resultValues, resultValues.size() * sizeof(cpx), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);
    }

    // free memory
    {
        err = hipFree(d_spanA); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_spanB); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_newSpan); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_connections); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_valuesA); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_valuesB); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_resultValues); cuda_err_check(err, __FILE__, __LINE__);
    }

    result.setValues(resultValues);
    return result;
}

struct gpuQtensor {
    unsigned char* span;
    cpx* values;
};

std::unordered_map<Contraction*, gpuQtensor> gpuQtensorMap;

hipblasHandle_t handle;

gpuQtensor moveQtensorToGPU (Contraction* contraction) {
    unsigned char* d_span;
    cpx* d_values;

    hipError_t err;

    err = hipMalloc(&d_span, contraction->data.span.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc(&d_values, contraction->data.values.size() * sizeof(cpx)); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(d_span, contraction->span.data(), contraction->data.span.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_values, contraction->data.values.data(), contraction->data.values.size() * sizeof(cpx), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    gpuQtensor gpuA = {d_span, d_values};
    return gpuA;
}

auto contractTreeGPU_r(Contraction* root) -> void {
    if (root == nullptr)
        return;
    if (root->kind == "C") {
        contractTreeGPU_r(root->left);
        contractTreeGPU_r(root->right);

        if (root->left->kind == "G") 
            gpuQtensorMap[root->left] = moveQtensorToGPU(root->left);

        if (root->right->kind == "G")
            gpuQtensorMap[root->right] = moveQtensorToGPU(root->right);

        std::vector<unsigned char> connections = findCommonValues(root->left->span, root->right->span);

        /** ----------------------------- CUDA ----------------------------- **/
        hipError_t err;

        // start transfering data to the GPU
        unsigned char *d_newSpan, *d_connections;
        cpx *d_resultValues;

        // memcopies
        {
            err = hipMalloc(&d_newSpan, root->span.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);
            err = hipMalloc(&d_connections, connections.size() * sizeof(unsigned char)); cuda_err_check(err, __FILE__, __LINE__);

            err = hipMalloc(&d_resultValues, (1 << (root->span.size()*2)) * sizeof(cpx)); cuda_err_check(err, __FILE__, __LINE__);

            err = hipMemcpy(d_newSpan, root->span.data(), root->span.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
            err = hipMemcpy(d_connections, connections.data(), connections.size() * sizeof(unsigned char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

            err = hipMemset(d_resultValues, 0, (1 << (root->span.size()*2)) * sizeof(cpx)); cuda_err_check(err, __FILE__, __LINE__);
        }

        gpuQtensorMap[root] = {d_newSpan, d_resultValues};

        // kernel call
        {
            int nels = 1 << (root->span.size()*2);
            int blocksize = 256;
            int numBlocks = round_div_up(nels, blocksize);

            // std::cout << "numBlocks: " << numBlocks << " blocksize: " << blocksize << std::endl;

            // if the span are the same use gemm
            if (root->left->span == root->right->span && false) {
                size_t nels = 1 << (root->span.size());
                cpx alpha = {1.0, 0.0};
                cpx beta = {0.0, 0.0};

                hipblasStatus_t status = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nels, nels, nels, &alpha, gpuQtensorMap[root->left].values, nels, gpuQtensorMap[root->right].values, nels, &beta, gpuQtensorMap[root].values, nels);
                if (status != HIPBLAS_STATUS_SUCCESS) {
                    fprintf(stderr, "hipblasCgemm failed: %s\n", _cudaGetErrorEnum(status));
                    exit(EXIT_FAILURE);
                }
                err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
                err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);
            }
            else {
                contractionKernel<<<numBlocks, blocksize>>>(gpuQtensorMap[root->right].span, gpuQtensorMap[root->left].span, gpuQtensorMap[root].span, d_connections, gpuQtensorMap[root->right].values, gpuQtensorMap[root->left].values, gpuQtensorMap[root].values, root->right->span.size(), root->left->span.size(), root->span.size(), connections.size());
                err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
                err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);
            }
            
            err = hipFree(gpuQtensorMap[root->left].span); cuda_err_check(err, __FILE__, __LINE__);
            err = hipFree(gpuQtensorMap[root->left].values); cuda_err_check(err, __FILE__, __LINE__);
            err = hipFree(gpuQtensorMap[root->right].span); cuda_err_check(err, __FILE__, __LINE__);
            err = hipFree(gpuQtensorMap[root->right].values); cuda_err_check(err, __FILE__, __LINE__);
            
        }
    }
}

auto contractTreeGPU(Contraction* root) -> void {
    hipblasStatus_t status;
    status = hipblasCreate(&handle); 
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "hipblasCreate failed: %s\n", _cudaGetErrorEnum(status));
        exit(EXIT_FAILURE);
    }

    contractTreeGPU_r(root);

    hipError_t err;
    std::vector<std::complex<float>> resultValues(1 << (root->span.size()*2));
    err = hipMemcpy(resultValues.data(), gpuQtensorMap[root].values, resultValues.size() * sizeof(cpx), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);
    root->data = QTensor();
    root->data.rank = root->span.size();
    root->data.setValues(resultValues);

    err = hipFree(gpuQtensorMap[root].span); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(gpuQtensorMap[root].values); cuda_err_check(err, __FILE__, __LINE__);
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "hipblasDestroy failed: %s\n", _cudaGetErrorEnum(status));
        exit(EXIT_FAILURE);
    }
}
