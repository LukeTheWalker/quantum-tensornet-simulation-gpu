#include "hip/hip_runtime.h"
#include "qTensor.cuh"
#include "qTensorCUDA.cuh"
#include "bitsetCU.cuh"
#include "Contraction.hpp"

#include <hip/hip_complex.h>
#include <unordered_map>
#include <hipblas.h>

#define DEBUG false

// using namespace cuda_classes;
#ifdef USE_FLOAT
using dtype = float;
using cpx = hipFloatComplex;
#else
using dtype = double;
using cpx = hipDoubleComplex;
#endif

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

auto findCommonValues = [](std::vector<unsigned char> set1, std::vector<unsigned char> set2) -> std::vector<unsigned char> {
    std::vector<unsigned char> commonValues;
    for (auto value : set1) {
        if (std::find(set2.begin(), set2.end(), value) != set2.end()) {
            commonValues.push_back((unsigned char)value);
        }
    }
    return commonValues;
};

unsigned char getIndexInSet(unsigned char* set, unsigned char element, size_t size) {
    for (size_t i = 0; i < size; i++) {
        if (set[i] == element) {
            return i;
        }
    }
    return 255; // Element not found in the set
}

__device__ void keepNtoMbits(cuda_classes::bitset& bits, size_t n, size_t m) 
{ 
    for (size_t i = 0; i < n; i++) 
    { 
        bits.set(i, 0);
    }  
    for (size_t i = m; i < 64; i++) 
    { 
        bits.set(i, 0);
    }  
}

__device__ void print_bitset(cuda_classes::bitset& bits) {
    for (size_t i = 0; i < 64; i++) {
        printf("%d", bits.get(i));
    }
    printf("\n");
}

__global__ void contractionKernel(cuda_classes::bitset* bit_addressesA, cuda_classes::bitset* bit_addressesB, cpx* d_valuesA, cpx* d_valuesB, cpx* d_resultValues, size_t rankA, size_t rankB, size_t rankResult, size_t connectionsSize, unsigned char* indexesA_connections, unsigned char* indexesB_connections)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= (1 << (rankResult*2))) return;

    #ifdef USE_FLOAT
    cpx value = hipCmulf(d_valuesA[bit_addressesA[i].to_ulong()], d_valuesB[bit_addressesB[i].to_ulong()]);
    d_resultValues[i] = hipCaddf(d_resultValues[i], value);
    #else
    cpx value = hipCmul(d_valuesA[bit_addressesA[i].to_ulong()], d_valuesB[bit_addressesB[i].to_ulong()]);
    d_resultValues[i] = hipCadd(d_resultValues[i], value);
    #endif
    
    size_t old_gray = 0;
    for (size_t m = 1; m < (1 << connectionsSize); m++)
    {
        size_t gray_code = m ^ (m >> 1);

        unsigned int position_vacant =  __ffsll(gray_code ^ old_gray) - 1;

        unsigned char indexA = indexesA_connections[position_vacant];
        unsigned char indexB = indexesB_connections[position_vacant];

        bit_addressesA[i].xor_op(1 << (rankA + indexA));
        bit_addressesB[i].xor_op(1 << (indexB));

        if (i == 0 && DEBUG) {
            // print indexA
            printf("IndexA: %d\n", indexA);
            // print indexB
            printf("IndexB: %d\n", indexB);
            printf("m: %d\n", m);
            printf("gray_code: %d\n", gray_code);
            printf("old_gray: %d\n", old_gray);
            printf("position_vacant: %d\n", position_vacant);
            printf("RankA: %d\n", rankA);
            printf("RankB: %d\n", rankB);
            printf("ConnectionsSize: %d\n", connectionsSize);
            printf("BitAdressA: ");
            print_bitset(bit_addressesA[i]);
            printf("BitAdressB: ");
            print_bitset(bit_addressesB[i]);
            printf("BitaAdressA: %d\n", bit_addressesA[i].to_ulong());
            printf("BitaAdressB: %d\n", bit_addressesB[i].to_ulong());
            printf("Values to multiply: %f + %fj and %f + %fj\n", d_valuesA[bit_addressesA[i].to_ulong()].x, d_valuesA[bit_addressesA[i].to_ulong()].y, d_valuesB[bit_addressesB[i].to_ulong()].x, d_valuesB[bit_addressesB[i].to_ulong()].y);
        }

        #ifdef USE_FLOAT
        cpx value = hipCmulf(d_valuesA[bit_addressesA[i].to_ulong()], d_valuesB[bit_addressesB[i].to_ulong()]);
        d_resultValues[i] = hipCaddf(d_resultValues[i], value);
        #else
        cpx value = hipCmul(d_valuesA[bit_addressesA[i].to_ulong()], d_valuesB[bit_addressesB[i].to_ulong()]);
        d_resultValues[i] = hipCadd(d_resultValues[i], value);
        #endif

        old_gray = gray_code;
    }
}

__global__ void compute_bit_address_map(cuda_classes::bitset* bit_addressesA, cuda_classes::bitset* bit_addressesB, size_t rankA, size_t rankB, size_t rankResult,  unsigned char* indexesA, unsigned char* indexesB){
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= (1 << (rankResult*2))) return;

    cuda_classes::bitset bits(i);


    for (size_t k = 0 ; k < rankResult; k++)
    {
        if (indexesB[k] != 255) bit_addressesB[i].set(rankB + indexesB[k], bits.get(rankResult + k));
        else                    bit_addressesA[i].set(rankA + indexesA[k], bits.get(rankResult + k));

        if (indexesA[k] != 255) bit_addressesA[i].set(indexesA[k], bits.get(k));
        else                    bit_addressesB[i].set(indexesB[k], bits.get(k));
    
    }

    if (i == 0 && DEBUG) {
        printf("BitAdressA: ");
        print_bitset(bit_addressesA[i]);
        printf("BitAdressB: ");
        print_bitset(bit_addressesB[i]);
    }
}

size_t round_div_up (size_t a, size_t b){
    return (a + b - 1)/b;
}

struct gpuQtensor {
    cpx* values;
};

std::unordered_map<Contraction*, gpuQtensor> gpuQtensorMap;

hipblasHandle_t handle;

gpuQtensor moveQtensorToGPU (Contraction* contraction, hipStream_t stream) {
    cpx* d_values;

    hipError_t err;

    err = hipMallocAsync(&d_values, contraction->data.getValuesSize() * sizeof(cpx), stream); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpyAsync(d_values,  contraction->data.values, contraction->data.getValuesSize() * sizeof(cpx), hipMemcpyHostToDevice, stream); cuda_err_check(err, __FILE__, __LINE__);

    gpuQtensor gpuA = {d_values};
    return gpuA;
}

hipEvent_t leftEvent, rightEvent;
auto contractTreeGPU_r(Contraction* root) -> void {
    if (root == nullptr)
        return;
    if (root->kind == "C") {
        contractTreeGPU_r(root->left);
        contractTreeGPU_r(root->right);

        hipError_t err;

        err = hipEventRecord(leftEvent, root->left->stream); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventRecord(rightEvent, root->right->stream); cuda_err_check(err, __FILE__, __LINE__);

        err = hipStreamWaitEvent(root->stream, leftEvent, 0); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamWaitEvent(root->stream, rightEvent, 0); cuda_err_check(err, __FILE__, __LINE__);

        err = hipStreamDestroy(root->left->stream); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamDestroy(root->right->stream); cuda_err_check(err, __FILE__, __LINE__);

        if (root->left->kind == "G") 
            gpuQtensorMap[root->left] = moveQtensorToGPU(root->left, root->stream);

        if (root->right->kind == "G")
            gpuQtensorMap[root->right] = moveQtensorToGPU(root->right, root->stream);

        std::vector<unsigned char> connections = findCommonValues(root->left->span, root->right->span);

        /** ----------------------------- CUDA ----------------------------- **/

        // start transfering data to the GPU
        cpx *d_resultValues;

        // unsigned char indexesA[root->span.size()];
        // unsigned char indexesB[root->span.size()];

        // unsigned char indexes_connectionsA[connections.size()];
        // unsigned char indexes_connectionsB[connections.size()];

        // use hipHostMalloc to allocate pinned memory
        unsigned char* indexesA, *indexesB, *indexes_connectionsA, *indexes_connectionsB;
        err = hipHostMalloc(&indexesA, root->span.size() * sizeof(unsigned char), hipHostMallocWriteCombined); cuda_err_check(err, __FILE__, __LINE__);
        err = hipHostMalloc(&indexesB, root->span.size() * sizeof(unsigned char), hipHostMallocWriteCombined); cuda_err_check(err, __FILE__, __LINE__);
        
        err = hipHostMalloc(&indexes_connectionsA, connections.size() * sizeof(unsigned char), hipHostMallocWriteCombined); cuda_err_check(err, __FILE__, __LINE__);
        err = hipHostMalloc(&indexes_connectionsB, connections.size() * sizeof(unsigned char), hipHostMallocWriteCombined); cuda_err_check(err, __FILE__, __LINE__);

        // memcopies
        {

            err = hipMallocAsync(&d_resultValues, (1 << (root->span.size()*2)) * sizeof(cpx), root->stream); cuda_err_check(err, __FILE__, __LINE__);

            err = hipMemsetAsync(d_resultValues, 0, (1 << (root->span.size()*2)) * sizeof(cpx), root->stream); cuda_err_check(err, __FILE__, __LINE__);
        }

        gpuQtensorMap[root] = {d_resultValues};

        // kernel call
        {
            size_t nels = 1 << (root->span.size()*2);
            size_t blocksize = 256;
            size_t numBlocks = round_div_up(nels, blocksize);
            size_t sharedMemSize = root->span.size() * sizeof(unsigned char);

            // std::cout << "numBlocks: " << numBlocks << " blocksize: " << blocksize << std::endl;

            // if the span are the same use gemm
            if (root->left->span == root->right->span) {
                hipblasSetStream(handle, root->stream);
                size_t nels = 1 << (root->span.size());
                cpx alpha = {1.0, 0.0};
                cpx beta = {0.0, 0.0};
                #ifdef USE_FLOAT
                hipblasStatus_t status = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nels, nels, nels, &alpha, gpuQtensorMap[root->left].values, nels, gpuQtensorMap[root->right].values, nels, &beta, gpuQtensorMap[root].values, nels);
                #else
                hipblasStatus_t status = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nels, nels, nels, &alpha, gpuQtensorMap[root->left].values, nels, gpuQtensorMap[root->right].values, nels, &beta, gpuQtensorMap[root].values, nels);
                #endif
                if (status != HIPBLAS_STATUS_SUCCESS) {
                    fprintf(stderr, "hipblasCgemm failed: %s\n", _cudaGetErrorEnum(status));
                    exit(EXIT_FAILURE);
                }
                // err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
                // err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);
            }
            else {
                cuda_classes::bitset* bit_addressesA, *bit_addressesB;

                err = hipMallocAsync(&bit_addressesA, nels * sizeof(cuda_classes::bitset), root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipMallocAsync(&bit_addressesB, nels * sizeof(cuda_classes::bitset), root->stream); cuda_err_check(err, __FILE__, __LINE__);

                err = hipMemsetAsync(bit_addressesA, 0, nels * sizeof(cuda_classes::bitset), root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipMemsetAsync(bit_addressesB, 0, nels * sizeof(cuda_classes::bitset), root->stream); cuda_err_check(err, __FILE__, __LINE__);

                #pragma omp parallel for
                for (size_t i = 0; i < root->span.size(); i++) {
                    indexesA[i] = getIndexInSet(root->left->span.data(), root->span[i], root->left->span.size());
                    indexesB[i] = getIndexInSet(root->right->span.data(),  root->span[i], root->right->span.size());
                }

                // // print indexA 
                // for (size_t i = 0; i < root->span.size(); i++) {
                //     std::cout << "IndexA[" << i << "]: " << (int)indexesA[i] << std::endl;
                // }

                // // print indexB
                // for (size_t i = 0; i < root->span.size(); i++) {
                //     std::cout << "IndexB[" << i << "]: " << (int)indexesB[i] << std::endl;
                // }
                
                #pragma omp parallel for
                for (size_t i = 0; i < connections.size(); i++) {
                    indexes_connectionsA[i] = getIndexInSet(root->left->span.data(), connections[i], root->left->span.size());
                    indexes_connectionsB[i] = getIndexInSet(root->right->span.data(),  connections[i], root->right->span.size());
                }

                unsigned char* d_indexesA, *d_indexesB;
                err = hipMallocAsync(&d_indexesA, root->span.size() * sizeof(unsigned char), root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipMallocAsync(&d_indexesB, root->span.size() * sizeof(unsigned char), root->stream); cuda_err_check(err, __FILE__, __LINE__);

                err = hipMemcpyAsync(d_indexesA, indexesA, root->span.size() * sizeof(unsigned char), hipMemcpyHostToDevice, root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipMemcpyAsync(d_indexesB, indexesB, root->span.size() * sizeof(unsigned char), hipMemcpyHostToDevice, root->stream); cuda_err_check(err, __FILE__, __LINE__);

                unsigned char* d_indexes_connectionsA, *d_indexes_connectionsB;
                err = hipMallocAsync(&d_indexes_connectionsA, connections.size() * sizeof(unsigned char), root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipMallocAsync(&d_indexes_connectionsB, connections.size() * sizeof(unsigned char), root->stream); cuda_err_check(err, __FILE__, __LINE__);

                err = hipMemcpyAsync(d_indexes_connectionsA, indexes_connectionsA, connections.size() * sizeof(unsigned char), hipMemcpyHostToDevice, root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipMemcpyAsync(d_indexes_connectionsB, indexes_connectionsB, connections.size() * sizeof(unsigned char), hipMemcpyHostToDevice, root->stream); cuda_err_check(err, __FILE__, __LINE__);

                double gb_used = (double)(sizeof(cuda_classes::bitset) * nels * 2) / (1024 * 1024 * 1024);

                if (gb_used > 1)
                    std::cout << "Memory allocation: " << (double)(sizeof(cuda_classes::bitset) * nels * 2) / (1024 * 1024 * 1024) << " GB" << std::endl;

                compute_bit_address_map<<<numBlocks, blocksize, 0, root->stream>>>(bit_addressesA, bit_addressesB, root->left->span.size(), root->right->span.size(), root->span.size(), d_indexesA, d_indexesB);

                // err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
                // err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

                contractionKernel<<<numBlocks, blocksize, 0, root->stream>>>(bit_addressesA, bit_addressesB, gpuQtensorMap[root->left].values, gpuQtensorMap[root->right].values, gpuQtensorMap[root].values, root->left->span.size(), root->right->span.size(), root->span.size(), connections.size(), d_indexes_connectionsA, d_indexes_connectionsB);

                // err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
                // err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

                err = hipFreeAsync(d_indexesA, root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipFreeAsync(d_indexesB, root->stream); cuda_err_check(err, __FILE__, __LINE__);

                err = hipFreeAsync(d_indexes_connectionsA, root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipFreeAsync(d_indexes_connectionsB, root->stream); cuda_err_check(err, __FILE__, __LINE__);

                err = hipFreeAsync(bit_addressesA, root->stream); cuda_err_check(err, __FILE__, __LINE__);
                err = hipFreeAsync(bit_addressesB, root->stream); cuda_err_check(err, __FILE__, __LINE__);
            }
            
            err = hipFreeAsync(gpuQtensorMap[root->left].values, root->stream); cuda_err_check(err, __FILE__, __LINE__);
            err = hipFreeAsync(gpuQtensorMap[root->right].values, root->stream); cuda_err_check(err, __FILE__, __LINE__);
        }
    }
}

auto contractTreeGPU(Contraction* root) -> void {
    hipblasStatus_t status;
    status = hipblasCreate(&handle); 
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "hipblasCreate failed: %s\n", _cudaGetErrorEnum(status));
        exit(EXIT_FAILURE);
    }
    
    hipError_t err;

    err = hipEventCreate(&leftEvent); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventCreate(&rightEvent); cuda_err_check(err, __FILE__, __LINE__);

    contractTreeGPU_r(root);

    err = hipEventDestroy(leftEvent); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventDestroy(rightEvent); cuda_err_check(err, __FILE__, __LINE__);

    std::vector<std::complex<dtype>> resultValues(1 << (root->span.size()*2));
    err = hipMemcpy(resultValues.data(), gpuQtensorMap[root].values, resultValues.size() * sizeof(cpx), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);
    root->data = QTensor();
    root->data.rank = root->span.size();
    root->data.setValues(resultValues);

    err = hipFree(gpuQtensorMap[root].values); cuda_err_check(err, __FILE__, __LINE__);
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "hipblasDestroy failed: %s\n", _cudaGetErrorEnum(status));
        exit(EXIT_FAILURE);
    }

    // std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;
}
